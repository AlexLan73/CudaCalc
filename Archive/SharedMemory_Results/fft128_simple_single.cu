/**
 * @file fft128_simple_single.cu
 * @brief FFT128 Simple version - 1 FFT per block
 * 
 * Честное сравнение с Shared Memory версией
 * 1 FFT128 на блок, 128 потоков на блок
 */

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cmath>

// FFT128 twiddle factors (64 values)
__constant__ float cos_table_128_single[64];
__constant__ float sin_table_128_single[64];

// Initialize twiddle factors for FFT128
void init_fft128_twiddles_single() {
    float h_cos[64], h_sin[64];
    
    for (int i = 0; i < 64; ++i) {
        float angle = -2.0f * M_PI * i / 128.0f;
        h_cos[i] = cosf(angle);
        h_sin[i] = sinf(angle);
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(cos_table_128_single), h_cos, 64 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(sin_table_128_single), h_sin, 64 * sizeof(float));
}

// 7-bit bit reversal
__device__ __forceinline__ int bitReverse7_single(int x) {
    int result = 0;
    #pragma unroll
    for (int i = 0; i < 7; ++i) {
        result = (result << 1) | (x & 1);
        x >>= 1;
    }
    return result;
}

__global__ void fft128_simple_single_kernel(const hipComplex* input, hipComplex* output, int num_windows) {
    // Block structure: 128 threads per block, 1 FFT per block
    const int fft_id = blockIdx.x;
    const int tid = threadIdx.x;  // 0-127
    
    if (fft_id >= num_windows) return;
    
    // Shared memory: 128 points (same as Shared Memory version)
    __shared__ hipComplex data[128];
    
    // Load input data with bit reversal
    const int input_idx = fft_id * 128 + tid;
    const int bit_rev_idx = bitReverse7_single(tid);
    data[bit_rev_idx] = input[input_idx];
    
    __syncthreads();
    
    // FFT128: 7 stages (точно такой же алгоритм как в Shared Memory версии)
    // Stage 1: m=2, m2=1, distance=1, 64 butterflies
    {
        const int m = 2;
        const int m2 = 1;
        const int group = tid / m;
        const int idx_in_group = tid % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128_single[tw_idx];
            const float tw_imag = sin_table_128_single[tw_idx];
            
            hipComplex a = data[idx1];
            hipComplex b = data[idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            data[idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            data[idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 2: m=4, m2=2, distance=2, 64 butterflies
    {
        const int m = 4;
        const int m2 = 2;
        const int group = tid / m;
        const int idx_in_group = tid % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128_single[tw_idx];
            const float tw_imag = sin_table_128_single[tw_idx];
            
            hipComplex a = data[idx1];
            hipComplex b = data[idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            data[idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            data[idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 3: m=8, m2=4, distance=4, 64 butterflies
    {
        const int m = 8;
        const int m2 = 4;
        const int group = tid / m;
        const int idx_in_group = tid % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128_single[tw_idx];
            const float tw_imag = sin_table_128_single[tw_idx];
            
            hipComplex a = data[idx1];
            hipComplex b = data[idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            data[idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            data[idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 4: m=16, m2=8, distance=8, 64 butterflies
    {
        const int m = 16;
        const int m2 = 8;
        const int group = tid / m;
        const int idx_in_group = tid % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128_single[tw_idx];
            const float tw_imag = sin_table_128_single[tw_idx];
            
            hipComplex a = data[idx1];
            hipComplex b = data[idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            data[idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            data[idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 5: m=32, m2=16, distance=16, 64 butterflies
    {
        const int m = 32;
        const int m2 = 16;
        const int group = tid / m;
        const int idx_in_group = tid % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128_single[tw_idx];
            const float tw_imag = sin_table_128_single[tw_idx];
            
            hipComplex a = data[idx1];
            hipComplex b = data[idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            data[idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            data[idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 6: m=64, m2=32, distance=32, 64 butterflies
    {
        const int m = 64;
        const int m2 = 32;
        const int group = tid / m;
        const int idx_in_group = tid % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128_single[tw_idx];
            const float tw_imag = sin_table_128_single[tw_idx];
            
            hipComplex a = data[idx1];
            hipComplex b = data[idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            data[idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            data[idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 7: m=128, m2=64, distance=64, 64 butterflies
    {
        const int m = 128;
        const int m2 = 64;
        const int group = tid / m;
        const int idx_in_group = tid % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128_single[tw_idx];
            const float tw_imag = sin_table_128_single[tw_idx];
            
            hipComplex a = data[idx1];
            hipComplex b = data[idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            data[idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            data[idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Store result
    const int output_idx = fft_id * 128 + tid;
    output[output_idx] = data[tid];
}

// Host launcher
extern "C" void launch_fft128_simple_single(const hipComplex* input, hipComplex* output, int num_windows) {
    // Initialize twiddle factors
    init_fft128_twiddles_single();
    
    dim3 block(128);  // 128 threads per block, 1 FFT per block
    dim3 grid(num_windows);  // One block per FFT
    
    fft128_simple_single_kernel<<<grid, block>>>(input, output, num_windows);
}
