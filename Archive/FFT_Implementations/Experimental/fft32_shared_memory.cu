/**
 * @file fft32_shared_memory.cu
 * @brief FFT32 GPU kernel using shared memory
 * 
 * Точная копия рабочего fft32_simple_correct.cu алгоритма,
 * адаптированная для shared memory
 */

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cmath>

// Pre-computed twiddle factors for FFT32 (копируем из рабочего алгоритма)
__constant__ float TWIDDLE_32_COS[16] = {
    1.000000f,   // k=0
    0.980785f,   // k=1
    0.923880f,   // k=2
    0.831470f,   // k=3
    0.707107f,   // k=4
    0.555570f,   // k=5
    0.382683f,   // k=6
    0.195090f,   // k=7
    0.000000f,   // k=8
   -0.195090f,   // k=9
   -0.382683f,   // k=10
   -0.555570f,   // k=11
   -0.707107f,   // k=12
   -0.831470f,   // k=13
   -0.923880f,   // k=14
   -0.980785f    // k=15
};

__constant__ float TWIDDLE_32_SIN[16] = {
    0.000000f,   // k=0
   -0.195090f,   // k=1
   -0.382683f,   // k=2
   -0.555570f,   // k=3
   -0.707107f,   // k=4
   -0.831470f,   // k=5
   -0.923880f,   // k=6
   -0.980785f,   // k=7
   -1.000000f,   // k=8
   -0.980785f,   // k=9
   -0.923880f,   // k=10
   -0.831470f,   // k=11
   -0.707107f,   // k=12
   -0.555570f,   // k=13
   -0.382683f,   // k=14
   -0.195090f    // k=15
};

// Bit reverse for 5 bits (копируем из рабочего алгоритма)
__device__ int bitReverse5_shared(int x) {
    int result = 0;
    result |= (x & 1) << 4;
    result |= (x & 2) << 2;
    result |= (x & 4);
    result |= (x & 8) >> 2;
    result |= (x & 16) >> 4;
    return result;
}

/**
 * FFT32 Shared Memory kernel - ТОЧНАЯ копия рабочего алгоритма
 * Block: 32 threads
 * Grid: num_windows blocks
 */
__global__ void fft32_shared_memory_kernel(
    const hipComplex* __restrict__ input,
    hipComplex* __restrict__ output,
    int num_windows
) {
    const int fft_id = blockIdx.x;
    const int tid = threadIdx.x;  // 0-31
    
    if (fft_id >= num_windows) return;
    
    __shared__ float2 data[32];
    
    // === STEP 1: Load with bit-reversal ===
    const int input_idx = fft_id * 32 + tid;
    const int reversed_tid = bitReverse5_shared(tid);
    data[reversed_tid] = make_float2(input[input_idx].x, input[input_idx].y);
    __syncthreads();
    
    // === STEP 2: Butterfly stages (5 stages for FFT32) ===
    for (int stage = 0; stage < 5; ++stage) {
        const int m = 1 << (stage + 1);  // 2, 4, 8, 16, 32
        const int m2 = m / 2;             // 1, 2, 4,  8, 16
        
        // Each thread works on pairs in parallel
        const int k = (tid / m2) * m;  // Group start
        const int j = tid % m2;         // Position in group
        
        if (tid < 16) {  // Only 16 threads work (rest wait)
            const int idx1 = k + j;
            const int idx2 = idx1 + m2;
            
            // Get twiddle from pre-computed table
            // W = W_32^((j * 32) / m)
            const int twiddle_idx = (j * 32) / m;
            const float tw_cos = TWIDDLE_32_COS[twiddle_idx];
            const float tw_sin = TWIDDLE_32_SIN[twiddle_idx];
            
            // Load values
            float2 u = data[idx1];
            float2 v = data[idx2];
            
            // t = W * v (complex multiply)
            float2 t;
            t.x = v.x * tw_cos - v.y * tw_sin;
            t.y = v.x * tw_sin + v.y * tw_cos;
            
            // Butterfly
            data[idx1] = make_float2(u.x + t.x, u.y + t.y);
            data[idx2] = make_float2(u.x - t.x, u.y - t.y);
        }
        __syncthreads();
    }
    
    // === STEP 3: Store result (NO shift!) ===
    const int output_idx = fft_id * 32 + tid;
    output[output_idx] = make_hipComplex(data[tid].x, data[tid].y);
}

// Host launcher
extern "C" void launch_fft32_shared_memory(
    const hipComplex* d_input,
    hipComplex* d_output,
    int num_windows
) {
    dim3 block(32);  // 32 threads per block
    dim3 grid(num_windows);  // One block per FFT
    
    fft32_shared_memory_kernel<<<grid, block>>>(d_input, d_output, num_windows);
}


