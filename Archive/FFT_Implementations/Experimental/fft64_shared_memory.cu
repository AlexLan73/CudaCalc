/**
 * @file fft64_shared_memory.cu
 * @brief FFT64 GPU kernel using shared memory
 * 
 * Точная копия рабочего fft64_simple_correct.cu алгоритма,
 * адаптированная для shared memory
 */

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cmath>

// FFT64 twiddle factors (32 values) - копируем из рабочего алгоритма
__constant__ float cos_table_64[32];
__constant__ float sin_table_64[32];

// Initialize twiddle factors for FFT64 (shared memory version)
void init_fft64_twiddles_shared() {
    float h_cos[32], h_sin[32];
    
    for (int i = 0; i < 32; ++i) {
        float angle = -2.0f * M_PI * i / 64.0f;
        h_cos[i] = cosf(angle);
        h_sin[i] = sinf(angle);
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(cos_table_64), h_cos, 32 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(sin_table_64), h_sin, 32 * sizeof(float));
}

// 6-bit bit reversal (копируем из рабочего алгоритма)
__device__ __forceinline__ int bitReverse6_shared(int x) {
    int result = 0;
    #pragma unroll
    for (int i = 0; i < 6; ++i) {
        result = (result << 1) | (x & 1);
        x >>= 1;
    }
    return result;
}

__global__ void fft64_shared_memory_kernel(const hipComplex* input, hipComplex* output, int num_windows) {
    // Block structure: [16, 64] = 1024 threads
    // threadIdx.x = 0-15 (which FFT window)
    // threadIdx.y = 0-63 (which point in FFT64)
    
    const int window_id = threadIdx.x;
    const int point_id = threadIdx.y;
    const int block_id = blockIdx.x;
    
    const int global_window = block_id * 16 + window_id;
    if (global_window >= num_windows) return;
    
    // Shared memory: [16 windows][64 points]
    extern __shared__ hipComplex shmem[];
    hipComplex (*window_data)[64] = (hipComplex (*)[64])shmem;
    
    // Load input data with bit reversal
    const int input_idx = global_window * 64 + point_id;
    const int bit_rev_idx = bitReverse6_shared(point_id);
    window_data[window_id][bit_rev_idx] = input[input_idx];
    
    __syncthreads();
    
    // FFT64: 6 stages (копируем точно из рабочего алгоритма)
    // Stage 1: m=2, m2=1, distance=1, 32 butterflies
    {
        const int m = 2;
        const int m2 = 1;
        const int group = point_id / m;  // which group (0-31)
        const int idx_in_group = point_id % m;  // position in group (0-1)
        
        if (idx_in_group < m2) {  // only first half of each group does work
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (64 / m);  // twiddle index
            const float tw_real = cos_table_64[tw_idx];
            const float tw_imag = sin_table_64[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 2: m=4, m2=2, distance=2, 32 butterflies
    {
        const int m = 4;
        const int m2 = 2;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (64 / m);
            const float tw_real = cos_table_64[tw_idx];
            const float tw_imag = sin_table_64[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 3: m=8, m2=4, distance=4, 32 butterflies
    {
        const int m = 8;
        const int m2 = 4;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (64 / m);
            const float tw_real = cos_table_64[tw_idx];
            const float tw_imag = sin_table_64[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 4: m=16, m2=8, distance=8, 32 butterflies
    {
        const int m = 16;
        const int m2 = 8;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (64 / m);
            const float tw_real = cos_table_64[tw_idx];
            const float tw_imag = sin_table_64[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 5: m=32, m2=16, distance=16, 32 butterflies
    {
        const int m = 32;
        const int m2 = 16;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (64 / m);
            const float tw_real = cos_table_64[tw_idx];
            const float tw_imag = sin_table_64[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 6: m=64, m2=32, distance=32, 32 butterflies
    {
        const int m = 64;
        const int m2 = 32;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (64 / m);
            const float tw_real = cos_table_64[tw_idx];
            const float tw_imag = sin_table_64[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Store result
    const int output_idx = global_window * 64 + point_id;
    output[output_idx] = window_data[window_id][point_id];
}

// Host launcher
extern "C" void launch_fft64_shared_memory(const hipComplex* input, hipComplex* output, int num_windows) {
    // Initialize twiddle factors
    init_fft64_twiddles_shared();
    
    dim3 block(16, 64);  // 16 windows, 64 points per window
    dim3 grid((num_windows + 15) / 16);  // Number of blocks needed
    
    // Shared memory size: 16 windows × 64 points × 8 bytes = 8192 bytes
    size_t shared_mem_size = 16 * 64 * sizeof(hipComplex);
    
    fft64_shared_memory_kernel<<<grid, block, shared_mem_size>>>(input, output, num_windows);
}
