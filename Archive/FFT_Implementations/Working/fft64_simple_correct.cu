#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cmath>

// FFT64 twiddle factors (32 values)
__constant__ float cos_table_64[32];
__constant__ float sin_table_64[32];

// Initialize twiddle factors for FFT64
void init_fft64_twiddles() {
    float h_cos[32], h_sin[32];
    
    for (int i = 0; i < 32; ++i) {
        float angle = -2.0f * M_PI * i / 64.0f;
        h_cos[i] = cosf(angle);
        h_sin[i] = sinf(angle);
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(cos_table_64), h_cos, 32 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(sin_table_64), h_sin, 32 * sizeof(float));
}

// 6-bit bit reversal
__device__ __forceinline__ int bitReverse6(int x) {
    int result = 0;
    #pragma unroll
    for (int i = 0; i < 6; ++i) {
        result = (result << 1) | (x & 1);
        x >>= 1;
    }
    return result;
}

__global__ void fft64_simple_kernel(const hipComplex* input, hipComplex* output, int num_windows) {
    // Block structure: [16, 64] = 1024 threads
    // threadIdx.x = 0-15 (which FFT window)
    // threadIdx.y = 0-63 (which point in FFT64)
    
    const int window_id = threadIdx.x;
    const int point_id = threadIdx.y;
    const int block_id = blockIdx.x;
    
    const int global_window = block_id * 16 + window_id;
    if (global_window >= num_windows) return;
    
    // Shared memory: [16 windows][64 points]
    extern __shared__ hipComplex shmem[];
    hipComplex (*window_data)[64] = (hipComplex (*)[64])shmem;
    
    // Load input data with bit reversal
    const int input_idx = global_window * 64 + point_id;
    const int bit_rev_idx = bitReverse6(point_id);
    window_data[window_id][bit_rev_idx] = input[input_idx];
    
    __syncthreads();
    
    // FFT64: 6 stages
    // Stage 1: m=2, m2=1, distance=1, 32 butterflies
    {
        const int m = 2;
        const int m2 = 1;
        const int group = point_id / m;  // which group (0-31)
        const int idx_in_group = point_id % m;  // position in group (0-1)
        
        if (idx_in_group < m2) {  // only first half of each group does work
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (64 / m);  // twiddle index
            const float tw_real = cos_table_64[tw_idx];
            const float tw_imag = sin_table_64[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 2: m=4, m2=2, distance=2, 32 butterflies
    {
        const int m = 4;
        const int m2 = 2;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (64 / m);
            const float tw_real = cos_table_64[tw_idx];
            const float tw_imag = sin_table_64[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 3: m=8, m2=4, distance=4, 32 butterflies
    {
        const int m = 8;
        const int m2 = 4;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (64 / m);
            const float tw_real = cos_table_64[tw_idx];
            const float tw_imag = sin_table_64[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 4: m=16, m2=8, distance=8, 32 butterflies
    {
        const int m = 16;
        const int m2 = 8;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (64 / m);
            const float tw_real = cos_table_64[tw_idx];
            const float tw_imag = sin_table_64[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 5: m=32, m2=16, distance=16, 32 butterflies
    {
        const int m = 32;
        const int m2 = 16;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (64 / m);
            const float tw_real = cos_table_64[tw_idx];
            const float tw_imag = sin_table_64[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 6: m=64, m2=32, distance=32, 32 butterflies
    if (point_id < 32) {
        const int idx1 = point_id;
        const int idx2 = point_id + 32;
        
        const int tw_idx = point_id;
        const float tw_real = cos_table_64[tw_idx];
        const float tw_imag = sin_table_64[tw_idx];
        
        hipComplex a = window_data[window_id][idx1];
        hipComplex b = window_data[window_id][idx2];
        
        const float b_tw_r = b.x * tw_real - b.y * tw_imag;
        const float b_tw_i = b.x * tw_imag + b.y * tw_real;
        
        window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
        window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
    }
    __syncthreads();
    
    // Store results
    const int output_idx = global_window * 64 + point_id;
    output[output_idx] = window_data[window_id][point_id];
}

extern "C" void launch_fft64_simple(const hipComplex* input, hipComplex* output, int num_windows) {
    static bool initialized = false;
    if (!initialized) {
        init_fft64_twiddles();
        initialized = true;
    }
    
    dim3 blockDim(16, 64);
    int num_blocks = (num_windows + 15) / 16;
    size_t shared_mem_size = 16 * 64 * sizeof(hipComplex);
    
    fft64_simple_kernel<<<num_blocks, blockDim, shared_mem_size>>>(
        input, output, num_windows
    );
}



