#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cmath>

// FFT1024 twiddle factors (512 values)
__constant__ float cos_table_1024[512];
__constant__ float sin_table_1024[512];

// Initialize twiddle factors for FFT1024
void init_fft1024_twiddles() {
    float h_cos[512], h_sin[512];
    
    for (int i = 0; i < 512; ++i) {
        float angle = -2.0f * M_PI * i / 1024.0f;
        h_cos[i] = cosf(angle);
        h_sin[i] = sinf(angle);
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(cos_table_1024), h_cos, 512 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(sin_table_1024), h_sin, 512 * sizeof(float));
}

// 10-bit bit reversal
__device__ __forceinline__ int bitReverse10(int x) {
    int result = 0;
    #pragma unroll
    for (int i = 0; i < 10; ++i) {
        result = (result << 1) | (x & 1);
        x >>= 1;
    }
    return result;
}

__global__ void fft1024_simple_kernel(const hipComplex* input, hipComplex* output, int num_windows) {
    // Block structure: [1, 1024] = 1024 threads
    // threadIdx.x = 0 (which FFT window)
    // threadIdx.y = 0-1023 (which point in FFT1024)
    
    const int window_id = threadIdx.x;
    const int point_id = threadIdx.y;
    const int block_id = blockIdx.x;
    
    const int global_window = block_id * 1 + window_id;
    if (global_window >= num_windows) return;
    
    // Shared memory: [1 window][1024 points]
    extern __shared__ hipComplex shmem[];
    hipComplex (*window_data)[1024] = (hipComplex (*)[1024])shmem;
    
    // Load input data with bit reversal
    const int input_idx = global_window * 1024 + point_id;
    const int bit_rev_idx = bitReverse10(point_id);
    window_data[window_id][bit_rev_idx] = input[input_idx];
    
    __syncthreads();
    
    // FFT1024: 10 stages
    // Stage 1: m=2, m2=1, distance=1, 512 butterflies
    {
        const int m = 2;
        const int m2 = 1;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (1024 / m);
            const float tw_real = cos_table_1024[tw_idx];
            const float tw_imag = sin_table_1024[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 2: m=4, m2=2, distance=2, 512 butterflies
    {
        const int m = 4;
        const int m2 = 2;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (1024 / m);
            const float tw_real = cos_table_1024[tw_idx];
            const float tw_imag = sin_table_1024[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 3: m=8, m2=4, distance=4, 512 butterflies
    {
        const int m = 8;
        const int m2 = 4;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (1024 / m);
            const float tw_real = cos_table_1024[tw_idx];
            const float tw_imag = sin_table_1024[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 4: m=16, m2=8, distance=8, 512 butterflies
    {
        const int m = 16;
        const int m2 = 8;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (1024 / m);
            const float tw_real = cos_table_1024[tw_idx];
            const float tw_imag = sin_table_1024[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 5: m=32, m2=16, distance=16, 512 butterflies
    {
        const int m = 32;
        const int m2 = 16;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (1024 / m);
            const float tw_real = cos_table_1024[tw_idx];
            const float tw_imag = sin_table_1024[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 6: m=64, m2=32, distance=32, 512 butterflies
    {
        const int m = 64;
        const int m2 = 32;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (1024 / m);
            const float tw_real = cos_table_1024[tw_idx];
            const float tw_imag = sin_table_1024[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 7: m=128, m2=64, distance=64, 512 butterflies
    {
        const int m = 128;
        const int m2 = 64;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (1024 / m);
            const float tw_real = cos_table_1024[tw_idx];
            const float tw_imag = sin_table_1024[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 8: m=256, m2=128, distance=128, 512 butterflies
    {
        const int m = 256;
        const int m2 = 128;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (1024 / m);
            const float tw_real = cos_table_1024[tw_idx];
            const float tw_imag = sin_table_1024[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 9: m=512, m2=256, distance=256, 512 butterflies
    {
        const int m = 512;
        const int m2 = 256;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (1024 / m);
            const float tw_real = cos_table_1024[tw_idx];
            const float tw_imag = sin_table_1024[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 10: m=1024, m2=512, distance=512, 512 butterflies
    if (point_id < 512) {
        const int idx1 = point_id;
        const int idx2 = point_id + 512;
        
        const int tw_idx = point_id;
        const float tw_real = cos_table_1024[tw_idx];
        const float tw_imag = sin_table_1024[tw_idx];
        
        hipComplex a = window_data[window_id][idx1];
        hipComplex b = window_data[window_id][idx2];
        
        const float b_tw_r = b.x * tw_real - b.y * tw_imag;
        const float b_tw_i = b.x * tw_imag + b.y * tw_real;
        
        window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
        window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
    }
    __syncthreads();
    
    // Store results
    const int output_idx = global_window * 1024 + point_id;
    output[output_idx] = window_data[window_id][point_id];
}

extern "C" void launch_fft1024_simple(const hipComplex* input, hipComplex* output, int num_windows) {
    static bool initialized = false;
    if (!initialized) {
        init_fft1024_twiddles();
        initialized = true;
    }
    
    dim3 blockDim(1, 1024);
    int num_blocks = num_windows; // 1 window per block
    size_t shared_mem_size = 1 * 1024 * sizeof(hipComplex);
    
    fft1024_simple_kernel<<<num_blocks, blockDim, shared_mem_size>>>(
        input, output, num_windows
    );
}


