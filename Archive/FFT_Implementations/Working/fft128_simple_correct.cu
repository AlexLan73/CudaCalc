#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cmath>

// FFT128 twiddle factors (64 values)
__constant__ float cos_table_128[64];
__constant__ float sin_table_128[64];

// Initialize twiddle factors for FFT128
void init_fft128_twiddles() {
    float h_cos[64], h_sin[64];
    
    for (int i = 0; i < 64; ++i) {
        float angle = -2.0f * M_PI * i / 128.0f;
        h_cos[i] = cosf(angle);
        h_sin[i] = sinf(angle);
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(cos_table_128), h_cos, 64 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(sin_table_128), h_sin, 64 * sizeof(float));
}

// 7-bit bit reversal
__device__ __forceinline__ int bitReverse7(int x) {
    int result = 0;
    #pragma unroll
    for (int i = 0; i < 7; ++i) {
        result = (result << 1) | (x & 1);
        x >>= 1;
    }
    return result;
}

__global__ void fft128_simple_kernel(const hipComplex* input, hipComplex* output, int num_windows) {
    // Block structure: [8, 128] = 1024 threads
    // threadIdx.x = 0-7 (which FFT window)
    // threadIdx.y = 0-127 (which point in FFT128)
    
    const int window_id = threadIdx.x;
    const int point_id = threadIdx.y;
    const int block_id = blockIdx.x;
    
    const int global_window = block_id * 8 + window_id;
    if (global_window >= num_windows) return;
    
    // Shared memory: [8 windows][128 points]
    extern __shared__ hipComplex shmem[];
    hipComplex (*window_data)[128] = (hipComplex (*)[128])shmem;
    
    // Load input data with bit reversal
    const int input_idx = global_window * 128 + point_id;
    const int bit_rev_idx = bitReverse7(point_id);
    window_data[window_id][bit_rev_idx] = input[input_idx];
    
    __syncthreads();
    
    // FFT128: 7 stages
    // Stage 1: m=2, m2=1, distance=1, 64 butterflies
    {
        const int m = 2;
        const int m2 = 1;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128[tw_idx];
            const float tw_imag = sin_table_128[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 2: m=4, m2=2, distance=2, 64 butterflies
    {
        const int m = 4;
        const int m2 = 2;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128[tw_idx];
            const float tw_imag = sin_table_128[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 3: m=8, m2=4, distance=4, 64 butterflies
    {
        const int m = 8;
        const int m2 = 4;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128[tw_idx];
            const float tw_imag = sin_table_128[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 4: m=16, m2=8, distance=8, 64 butterflies
    {
        const int m = 16;
        const int m2 = 8;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128[tw_idx];
            const float tw_imag = sin_table_128[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 5: m=32, m2=16, distance=16, 64 butterflies
    {
        const int m = 32;
        const int m2 = 16;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128[tw_idx];
            const float tw_imag = sin_table_128[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 6: m=64, m2=32, distance=32, 64 butterflies
    {
        const int m = 64;
        const int m2 = 32;
        const int group = point_id / m;
        const int idx_in_group = point_id % m;
        
        if (idx_in_group < m2) {
            const int idx1 = group * m + idx_in_group;
            const int idx2 = idx1 + m2;
            
            const int tw_idx = idx_in_group * (128 / m);
            const float tw_real = cos_table_128[tw_idx];
            const float tw_imag = sin_table_128[tw_idx];
            
            hipComplex a = window_data[window_id][idx1];
            hipComplex b = window_data[window_id][idx2];
            
            const float b_tw_r = b.x * tw_real - b.y * tw_imag;
            const float b_tw_i = b.x * tw_imag + b.y * tw_real;
            
            window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
            window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
        }
    }
    __syncthreads();
    
    // Stage 7: m=128, m2=64, distance=64, 64 butterflies
    if (point_id < 64) {
        const int idx1 = point_id;
        const int idx2 = point_id + 64;
        
        const int tw_idx = point_id;
        const float tw_real = cos_table_128[tw_idx];
        const float tw_imag = sin_table_128[tw_idx];
        
        hipComplex a = window_data[window_id][idx1];
        hipComplex b = window_data[window_id][idx2];
        
        const float b_tw_r = b.x * tw_real - b.y * tw_imag;
        const float b_tw_i = b.x * tw_imag + b.y * tw_real;
        
        window_data[window_id][idx1] = make_hipComplex(a.x + b_tw_r, a.y + b_tw_i);
        window_data[window_id][idx2] = make_hipComplex(a.x - b_tw_r, a.y - b_tw_i);
    }
    __syncthreads();
    
    // Store results
    const int output_idx = global_window * 128 + point_id;
    output[output_idx] = window_data[window_id][point_id];
}

extern "C" void launch_fft128_simple(const hipComplex* input, hipComplex* output, int num_windows) {
    static bool initialized = false;
    if (!initialized) {
        init_fft128_twiddles();
        initialized = true;
    }
    
    dim3 blockDim(8, 128);
    int num_blocks = (num_windows + 7) / 8;
    size_t shared_mem_size = 8 * 128 * sizeof(hipComplex);
    
    fft128_simple_kernel<<<num_blocks, blockDim, shared_mem_size>>>(
        input, output, num_windows
    );
}


